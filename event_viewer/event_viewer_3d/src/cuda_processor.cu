#include "hip/hip_runtime.h"
#include "cuda_processor.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>


#define CUDA_CHECK(err) { \
    hipError_t err_ = (err); \
    if (err_ != hipSuccess) { \
        std::cerr << "CUDA Error in " << __FILE__ << " at line " << __LINE__ \
                  << ": " << hipGetErrorString(err_) << std::endl; \
        exit(EXIT_FAILURE); \
    } \
}

static struct hipGraphicsResource* vbo_resource_cu = nullptr;

void register_gl_buffer(GLuint vbo) {
    CUDA_CHECK(hipGraphicsGLRegisterBuffer(&vbo_resource_cu, vbo, hipGraphicsRegisterFlagsWriteDiscard));
}
void unregister_gl_buffer() {
    if (vbo_resource_cu) {
        CUDA_CHECK(hipGraphicsUnregisterResource(vbo_resource_cu));
        vbo_resource_cu = nullptr;
    }
}
void init_cuda_for_gl() {
    CUDA_CHECK(hipSetDevice(0));
    CUDA_CHECK(hipFree(0));
    std::cout << "CUDA initialized for OpenGL Interop on Device 0." << std::endl;
}

__global__ void events_to_vertices(const EventCD* d_in, Vertex* d_out, int total_events, unsigned int* d_count, int width, int height, int64_t t_offset, double base_time, float3 color_on, float3 color_off) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= total_events) return;

    unsigned int write_idx = atomicAdd(d_count, 1);

    EventCD event = d_in[idx];
    Vertex v;
    // ... 座標計算は同じ ...
    v.x = (static_cast<float>(event.x) / static_cast<float>(width) - 0.5f) * 2.0f;
    v.y = (static_cast<float>(event.y) / static_cast<float>(height) - 0.5f) * -2.0f;
    double absolute_time = static_cast<double>(t_offset) + event.t;
    v.z = static_cast<float>(absolute_time - base_time);

    // ★★★ 色を設定から適用 ★★★
    // 0.0-1.0のfloatを0-255のuint8_tに変換
    if (event.pol == 1) {
        v.r = static_cast<uint8_t>(color_on.x * 255.0f);
        v.g = static_cast<uint8_t>(color_on.y * 255.0f);
        v.b = static_cast<uint8_t>(color_on.z * 255.0f);
    } else {
        v.r = static_cast<uint8_t>(color_off.x * 255.0f);
        v.g = static_cast<uint8_t>(color_off.y * 255.0f);
        v.b = static_cast<uint8_t>(color_off.z * 255.0f);
    }
    v.a = 255;
    
    d_out[write_idx] = v;
}
// ★★★ process_all_events関数も t_offset と base_time を受け取るように修正 ★★★
unsigned int process_all_events(const std::vector<EventCD>& all_events, int width, int height, int64_t t_offset, double base_time, const ColorConfig& colors) {
    if (all_events.empty() || !vbo_resource_cu) return 0;
    
    std::cout << "--- 全イベントのCUDA処理を開始..." << std::endl;
    EventCD* d_events = nullptr;
    size_t data_size = all_events.size() * sizeof(EventCD);
    CUDA_CHECK(hipMalloc(&d_events, data_size));

    std::cout << "--- CPUからGPUへのデータ転送を開始 (" 
              << data_size / (1024 * 1024) << " MB)... ---" << std::endl;
    CUDA_CHECK(hipMemcpy(d_events, all_events.data(), data_size, hipMemcpyHostToDevice));
    std::cout << "--- データ転送完了。カーネルを実行します ---" << std::endl;

    unsigned int* d_count = nullptr;
    CUDA_CHECK(hipMalloc(&d_count, sizeof(unsigned int)));
    CUDA_CHECK(hipMemset(d_count, 0, sizeof(unsigned int)));

    Vertex* d_vbo_ptr = nullptr;
    CUDA_CHECK(hipGraphicsMapResources(1, &vbo_resource_cu, 0));
    CUDA_CHECK(hipGraphicsResourceGetMappedPointer((void**)&d_vbo_ptr, nullptr, vbo_resource_cu));

    int threads = 256;
    int blocks = (all_events.size() + threads - 1) / threads;
    
    // ★★★ glm::vec3 から float3 に変換 ★★★
    float3 color_on = make_float3(colors.event_on.r, colors.event_on.g, colors.event_on.b);
    float3 color_off = make_float3(colors.event_off.r, colors.event_off.g, colors.event_off.b);

    // ★★★ カーネル呼び出し時に色情報を渡す ★★★
    events_to_vertices<<<blocks, threads>>>(d_events, d_vbo_ptr, all_events.size(), d_count, width, height, t_offset, base_time, color_on, color_off);
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipGraphicsUnmapResources(1, &vbo_resource_cu, 0));

    unsigned int final_count = 0;
    CUDA_CHECK(hipMemcpy(&final_count, d_count, sizeof(unsigned int), hipMemcpyDeviceToHost));
    
    CUDA_CHECK(hipFree(d_count));
    CUDA_CHECK(hipFree(d_events));

    std::cout << "--- CUDA処理完了: " << final_count << "個の頂点を生成 ---" << std::endl;
    return final_count;
}