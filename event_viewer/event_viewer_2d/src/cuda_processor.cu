#include "hip/hip_runtime.h"
#include "cuda_processor.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

// 色の設定
__device__ const unsigned char POLARITY_1_R = 255, POLARITY_1_G = 50, POLARITY_1_B = 50;
__device__ const unsigned char POLARITY_0_R = 50, POLARITY_0_G = 50, POLARITY_0_B = 255;


#define CUDA_CHECK(err) { \
    hipError_t err_ = (err); \
    if (err_ != hipSuccess) { \
        std::cerr << "CUDA Error in " << __FILE__ << " at line " << __LINE__ \
                  << ": " << hipGetErrorString(err_) << std::endl; \
        exit(EXIT_FAILURE); \
    } \
}

static struct hipGraphicsResource* vbo_resource_cu = nullptr;

void register_gl_buffer(GLuint vbo) {
    CUDA_CHECK(hipGraphicsGLRegisterBuffer(&vbo_resource_cu, vbo, hipGraphicsRegisterFlagsWriteDiscard));
}
void unregister_gl_buffer() {
    if (vbo_resource_cu) {
        CUDA_CHECK(hipGraphicsUnregisterResource(vbo_resource_cu));
        vbo_resource_cu = nullptr;
    }
}
void init_cuda_for_gl() {
    CUDA_CHECK(hipSetDevice(0));
    CUDA_CHECK(hipFree(0));
    std::cout << "CUDA initialized for OpenGL Interop on Device 0." << std::endl;
}

// ★★★ カーネルが t_offset と base_time を受け取るように変更 ★★★
__global__ void events_to_vertices(const EventCD* d_in, Vertex* d_out, int total_events, unsigned int* d_count, int width, int height, int64_t t_offset, double base_time) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= total_events) return;

    unsigned int write_idx = atomicAdd(d_count, 1);
    
    EventCD event = d_in[idx];
    Vertex v;
    v.x = (static_cast<float>(event.x) / static_cast<float>(width) - 0.5f) * 2.0f; 
    v.y = (static_cast<float>(event.y) / static_cast<float>(height) - 0.5f) * -2.0f;
    
    // ★★★ 絶対時刻を計算し、そこから基準時刻を引いて相対的なZ座標を計算 ★★★
    double absolute_time = static_cast<double>(t_offset) + event.t;
    v.z = static_cast<float>(absolute_time - base_time);

    if (event.p == 1) {
        v.r = POLARITY_1_R; v.g = POLARITY_1_G; v.b = POLARITY_1_B; v.a = 255;
    } else {
        v.r = POLARITY_0_R; v.g = POLARITY_0_G; v.b = POLARITY_0_B; v.a = 255;
    }
    
    d_out[write_idx] = v;
}

// ★★★ process_all_events関数も t_offset と base_time を受け取るように修正 ★★★
unsigned int process_all_events(const std::vector<EventCD>& all_events, int width, int height, int64_t t_offset, double base_time) {
    if (all_events.empty() || !vbo_resource_cu) return 0;
    
    std::cout << "--- 全イベントのCUDA処理を開始..." << std::endl;
    EventCD* d_events = nullptr;
    size_t data_size = all_events.size() * sizeof(EventCD);
    CUDA_CHECK(hipMalloc(&d_events, data_size));

    std::cout << "--- CPUからGPUへのデータ転送を開始 (" 
              << data_size / (1024 * 1024) << " MB)... ---" << std::endl;
    CUDA_CHECK(hipMemcpy(d_events, all_events.data(), data_size, hipMemcpyHostToDevice));
    std::cout << "--- データ転送完了。カーネルを実行します ---" << std::endl;

    unsigned int* d_count = nullptr;
    CUDA_CHECK(hipMalloc(&d_count, sizeof(unsigned int)));
    CUDA_CHECK(hipMemset(d_count, 0, sizeof(unsigned int)));

    Vertex* d_vbo_ptr = nullptr;
    CUDA_CHECK(hipGraphicsMapResources(1, &vbo_resource_cu, 0));
    CUDA_CHECK(hipGraphicsResourceGetMappedPointer((void**)&d_vbo_ptr, nullptr, vbo_resource_cu));

    int threads = 256;
    int blocks = (all_events.size() + threads - 1) / threads;
    
    // ★★★ カーネル呼び出し時に t_offset と base_time を渡す ★★★
    events_to_vertices<<<blocks, threads>>>(d_events, d_vbo_ptr, all_events.size(), d_count, width, height, t_offset, base_time);
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipGraphicsUnmapResources(1, &vbo_resource_cu, 0));

    unsigned int final_count = 0;
    CUDA_CHECK(hipMemcpy(&final_count, d_count, sizeof(unsigned int), hipMemcpyDeviceToHost));
    
    CUDA_CHECK(hipFree(d_count));
    CUDA_CHECK(hipFree(d_events));

    std::cout << "--- CUDA処理完了: " << final_count << "個の頂点を生成 ---" << std::endl;
    return final_count;
}